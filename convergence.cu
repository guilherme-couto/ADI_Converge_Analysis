#include "hip/hip_runtime.h"
#include "./src/methods.h"

int main(int argc, char *argv[])
{
    // Parameters
    char *method;
    real delta_t;
    real delta_x;
    real theta;

    // Read parameters
    if (argc != 5)
    {
        printf("Usage: %s method delta_t delta_x theta\n", argv[0]);
        return 1;
    }
    method = argv[1];
    delta_t = atof(argv[2]);
    delta_x = atof(argv[3]);
    theta = atof(argv[4]);

    // Call function
    runSimulation(method, delta_t, delta_x, theta);

    return 0;

    // TESTES THOMAS ///////////////////////////////////////////////////////////////////////////////////////////////////////
    // int N = 3;

    // real **RHS = (real **)malloc(N * sizeof(real *));
    // real *c_prime = (real *)malloc(N * sizeof(real));   // aux for Thomas
    // real *d_prime = (real *)malloc(N * sizeof(real));   // aux for Thomas
    // for (int i = 0; i < N; i++)
    // {
    //     RHS[i] = (real *)malloc(N * sizeof(real));
    // }

    // // Populate RHS
    // RHS[0][0] = 13;
    // RHS[0][1] = 14;
    // RHS[0][2] = 15;
    // RHS[1][0] = 4;
    // RHS[1][1] = 5;
    // RHS[1][2] = 6;
    // RHS[2][0] = -5;
    // RHS[2][1] = -4;
    // RHS[2][2] = -3;

    // printf("\nMatriz RHS\n");
    // for (int i = 0; i < N; i++)
    // {
    //     for (int j = 0; j < N; j++)
    //     {
    //         printf("%lf ", RHS[i][j]);
    //     }
    //     printf("\n");
    // }
    // printf("\n");

    // // Auxiliary arrays for Thomas algorithm
    // real *la = (real *)malloc(N * sizeof(real));
    // real *lb = (real *)malloc(N * sizeof(real));
    // real *lc = (real *)malloc(N * sizeof(real));
    // real *d = (real *)malloc(N * sizeof(real));
    // real *solution = (real *)malloc(N * sizeof(real));

    // populateDiagonalThomasAlgorithm(la, lb, lc, N, 2);

    // for (int i = 0; i < N; i++)
    //     thomasAlgorithm(la, lb, lc, c_prime, d_prime, N, RHS[i]);

    // printf("\nResultado do Thomas para diff imp x\n");
    // for (int i = 0; i < N; i++)
    // {
    //     for (int j = 0; j < N; j++)
    //     {
    //         printf("%lf ", RHS[i][j]);
    //     }
    //     printf("\n");
    // }
    // printf("\n");

    // for (int j = 0; j < N; j++)
    // {
    //     copyColumnToVector(RHS, d, N, j);
    //     thomasAlgorithm(la, lb, lc, c_prime, d_prime, N, d);
    //     copyVectorToColumn(RHS, d, N, j);
    // }

    // printf("\nResultado do Thomas para diff imp y\n");
    // for (int i = 0; i < N; i++)
    // {
    //     for (int j = 0; j < N; j++)
    //     {
    //         printf("%lf ", RHS[i][j]);
    //     }
    //     printf("\n");
    // }
    // printf("\n");

    // // Free memory from host
    // for (int i = 0; i < N; i++)
    //     free(RHS[i]);
    // free(RHS);
    // free(la);
    // free(lb);
    // free(lc);
    // free(c_prime);
    // free(d_prime);
    // free(d);
    // free(solution);

    // return 0;
    // /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    
    // TESTE PARALELO /////////////////////////////////////////////////////////////////////////////////////////
    // int N = 3;

    // real *RHS;
    // RHS = (real *)malloc(N * N * sizeof(real));
    // RHS[0]=13;
    // RHS[1]=14;
    // RHS[2]=15;
    // RHS[3]=4;
    // RHS[4]=5;
    // RHS[5]=6;
    // RHS[6]=-5;
    // RHS[7]=-4;
    // RHS[8]=-3;

    // // Auxiliary arrays for Thomas algorithm
    // real *la = (real *)malloc(N * sizeof(real));
    // real *lb = (real *)malloc(N * sizeof(real));
    // real *lc = (real *)malloc(N * sizeof(real));

    
    // populateDiagonalThomasAlgorithm(la, lb, lc, N, 2);

    // // Prefactorization
    // thomasFactorConstantBatch(la, lb, lc, N);

    // real *d_RHS, *d_aux;
    // real *d_la, *d_lb, *d_lc;
    // hipError_t cudaStatus1, cudaStatus2, cudaStatus3, cudaStatus6, cudaStatus7, cudaStatus8;
    
    // cudaStatus2 = hipMalloc(&d_RHS, N * N * sizeof(real));
    // cudaStatus2 = hipMalloc(&d_aux, N * N * sizeof(real));
    // cudaStatus6 = hipMalloc(&d_la, N * sizeof(real));
    // cudaStatus7 = hipMalloc(&d_lb, N * sizeof(real));
    // cudaStatus8 = hipMalloc(&d_lc, N * sizeof(real));
    // if (cudaStatus2 != hipSuccess || cudaStatus6 != hipSuccess || cudaStatus7 != hipSuccess || cudaStatus8 != hipSuccess)
    // {
    //     printf("hipMalloc failed!\n");
    //     exit(EXIT_FAILURE);
    // }

    // // Copy memory from host to device of the matrices (2D arrays)
    // cudaStatus1 = hipMemcpy(d_RHS, RHS, N * N * sizeof(real), hipMemcpyHostToDevice);
    // if (cudaStatus1 != hipSuccess)
    // {
    //     printf("hipMemcpy failed!\n");
    //     exit(EXIT_FAILURE);
    // }

    // // Copy memory of diagonals from host to device
    // cudaStatus1 = hipMemcpy(d_la, la, N * sizeof(real), hipMemcpyHostToDevice);
    // cudaStatus2 = hipMemcpy(d_lb, lb, N * sizeof(real), hipMemcpyHostToDevice);
    // cudaStatus3 = hipMemcpy(d_lc, lc, N * sizeof(real), hipMemcpyHostToDevice);
    // if (cudaStatus1 != hipSuccess || cudaStatus2 != hipSuccess || cudaStatus3 != hipSuccess)
    // {
    //     printf("hipMemcpy failed!\n");
    //     exit(EXIT_FAILURE);
    // }
    // printf("All cudaMallocs done!\n");

    // // Block and grid size
    // // For parallel Thomas
    // printf("N = %d\n", N);
    // int numBlocks = N / 100;
    // if (numBlocks == 0)
    //     numBlocks = 1;
    // int blockSize = round(N / numBlocks) + 1;
        
    // if (blockSize % 32 != 0)
    //     blockSize = 32 * ((blockSize / 32) + 1);

    // // For other kernels
    // int GRID_SIZE = ceil((N*N*1.0) / (BLOCK_SIZE*1.0));
    // if (GRID_SIZE == 0)
    //     GRID_SIZE = 1;

    // printf("For 1st Part and Transpose -> Grid size %d, Block size %d\n", GRID_SIZE, BLOCK_SIZE);
    // printf("Total for 1st Part and Transpose: %d\n", GRID_SIZE*BLOCK_SIZE);
    // printf("For Thomas Algorithm -> Grid size: %d, Block size: %d\n", numBlocks, blockSize);
    // printf("Total for Thomas Algorithm: %d\n", numBlocks*blockSize);

    // cuThomasConstantBatch<<<numBlocks, blockSize>>>(d_la, d_lb, d_lc, d_RHS, N);
    // hipDeviceSynchronize();

    // //Copy d_RHS to RHS
    // cudaStatus1 = hipMemcpy(RHS, d_RHS, N * N * sizeof(real), hipMemcpyDeviceToHost);
    // if (cudaStatus1 != hipSuccess)
    // {
    //     printf("hipMemcpy failed!\n");
    //     exit(EXIT_FAILURE);
    // }

    // printf("\nResultado sem transpor a matriz RHS\n");
    // for (int i = 0; i < N; i++)
    // {
    //     for (int j = 0; j < N; j++)
    //     {
    //         int index = i * N + j;
    //         printf("%lf ", RHS[index]);
    //     }
    //     printf("\n");
    // }

    // //########################
    // RHS[0]=13;
    // RHS[1]=14;
    // RHS[2]=15;
    // RHS[3]=4;
    // RHS[4]=5;
    // RHS[5]=6;
    // RHS[6]=-5;
    // RHS[7]=-4;
    // RHS[8]=-3;

    // cudaStatus1 = hipMemcpy(d_RHS, RHS, N * N * sizeof(real), hipMemcpyHostToDevice);

    // transposeDiagonalCol<<<GRID_SIZE, BLOCK_SIZE>>>(d_RHS, d_aux, N);
    // hipDeviceSynchronize();

    // //Copy d_RHS to RHS
    // cudaStatus1 = hipMemcpy(RHS, d_aux, N * N * sizeof(real), hipMemcpyDeviceToHost);
    
    // printf("\nRHS transposta (interleaved)\n");
    // for (int i = 0; i < N; i++)
    // {
    //     for (int j = 0; j < N; j++)
    //     {
    //         int index = i * N + j;
    //         printf("%lf ", RHS[index]);
    //     }
    //     printf("\n");
    // }

    // cuThomasConstantBatch<<<numBlocks, blockSize>>>(d_la, d_lb, d_lc, d_aux, N);
    // hipDeviceSynchronize();

    // //Copy d_RHS to RHS
    // cudaStatus1 = hipMemcpy(RHS, d_aux, N * N * sizeof(real), hipMemcpyDeviceToHost);
    // if (cudaStatus1 != hipSuccess)
    // {
    //     printf("hipMemcpy failed!\n");
    //     exit(EXIT_FAILURE);
    // }

    // printf("\nResultado transpondo a matriz RHS\n");
    // for (int i = 0; i < N; i++)
    // {
    //     for (int j = 0; j < N; j++)
    //     {
    //         int index = i * N + j;
    //         printf("%lf ", RHS[index]);
    //     }
    //     printf("\n");
    // }

    // transposeDiagonalCol<<<GRID_SIZE, BLOCK_SIZE>>>(d_aux, d_RHS, N);
    // hipDeviceSynchronize();

    // //Copy d_RHS to RHS
    // cudaStatus1 = hipMemcpy(RHS, d_RHS, N * N * sizeof(real), hipMemcpyDeviceToHost);
    
    // printf("\n2a RHS transposta (interleaved)\n");
    // for (int i = 0; i < N; i++)
    // {
    //     for (int j = 0; j < N; j++)
    //     {
    //         int index = i * N + j;
    //         printf("%lf ", RHS[index]);
    //     }
    //     printf("\n");
    // }

    // cuThomasConstantBatch<<<numBlocks, blockSize>>>(d_la, d_lb, d_lc, d_RHS, N);
    // hipDeviceSynchronize();

    // //Copy d_RHS to RHS
    // cudaStatus1 = hipMemcpy(RHS, d_RHS, N * N * sizeof(real), hipMemcpyDeviceToHost);
    // if (cudaStatus1 != hipSuccess)
    // {
    //     printf("hipMemcpy failed!\n");
    //     exit(EXIT_FAILURE);
    // }

    // printf("\nResultado Pós segundo Sistema\n");
    // for (int i = 0; i < N; i++)
    // {
    //     for (int j = 0; j < N; j++)
    //     {
    //         int index = i * N + j;
    //         printf("%lf ", RHS[index]);
    //     }
    //     printf("\n");
    // }



    // // Free memory from host
    // free(RHS);
    // free(la);
    // free(lb);
    // free(lc);

    // // Free memory from device
    // hipFree(d_RHS);
    // hipFree(d_aux);
    // hipFree(d_la);
    // hipFree(d_lb);
    // hipFree(d_lc);
    // return 0;
    
}