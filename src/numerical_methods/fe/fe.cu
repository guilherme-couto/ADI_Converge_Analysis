#include "hip/hip_runtime.h"
#include "../numerical_methods.h"
#include "../numerical_methods_helpers.h"
#include "../../cell_models/cell_models.h"

static __global__ void solveAndUpdateSV(const int Nx, const int Ny, const real delta_t, const real phi_x, const real phi_y,
                                        const real diff_coeff, const real actualTime, const int num_active_stimuli,
                                        const Stimulus *d_active_stimuli, const real *d_Vm, real *d_sV, real *d_RHS, const CellModel cell_model)
{
    // Obtain the thread index
    const int i = blockIdx.y * blockDim.y + threadIdx.y;
    const int j = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx = i * Nx + j;

    if (i < Ny && j < Nx)
    {
        // Declare auxiliary array
        real d_actualsV[MAX_NSV];

        // Calculate the explicit part of the RHS, including the diffusion term in both directions
        real actualVm = d_Vm[idx];
        select_get_actual_sV(cell_model, d_actualsV, d_sV, idx);
        
        // Stimulation
        real stim = get_stimulus_value(actualTime, i, j, d_active_stimuli, num_active_stimuli);

        // Update variables explicitly
        real diff_term = compute_diffusion_term(d_Vm, i, j, Nx, Ny, diff_coeff, phi_x, phi_y);

        d_RHS[idx] = actualVm + diff_term + delta_t * (stim - select_compute_dVmdt(cell_model, actualVm, d_actualsV));

        // Update state variables
        select_update_sV(cell_model, d_sV, d_actualsV, actualVm, d_actualsV, delta_t, idx);
    }
}

void runFE_CUDA(const SimulationConfig *config, Measurement *measurement, const real *time_array,
                const CellModelSolver *cell_model_solver, real *Vm, real *sV)
{
    // Unpack configuration parameters
    const int M = config->M;
    const int Nx = config->Nx;
    const int Ny = config->Ny;
    const real delta_t = config->dt;
    const real delta_x = config->dx;
    const real delta_y = config->dy;
    const real sigma = config->sigma;
    const int numberOfStimuli = config->stimulus_count;
    const Stimulus *stimuli = config->stimuli;

    const CellModel cell_model = config->cell_model;
    const bool saveFrames = config->save_frames;
    const int frameSaveRate = config->frame_save_rate;
    const char *pathToSaveData = config->output_dir;
    const char *file_extension = config->file_extension;
    const save_function_t save_function = config->save_function;
    const bool measureVelocity = config->measure_velocity;
    
    // Get the solver functions
    const real activation_threshold = cell_model_solver->activation_thershold;
    const compute_diffusion_coefficient_t compute_diffusion_coefficient = cell_model_solver->compute_diffusion_coefficient;
    
    // Measure velocity variables
    real stim_velocity, t0, t1;
    const real x0 = config->Lx / 3.0f;
    const real x1 = 2.0f * x0;
    const int idx_x0 = round(x0 / delta_x) + 1;
    const int idx_x1 = round(x1 / delta_x) + 1;
    bool aux_stim_velocity_flag = false;
    bool stim_velocity_measured = false;

    // Auxiliary variables for the loops
    int timeStepCounter = 0;
    real actualTime = 0.0f;
    
    // Create device variables, allocate memory on device, and copy data
    real *d_Vm, *d_sV;
    Stimulus *d_stimuli;
    const int total_points = Nx * Ny;

    CUDA_CALL(hipMalloc(&d_Vm, total_points * sizeof(real)));
    CUDA_CALL(hipMalloc(&d_sV, total_points * cell_model_solver->n_state_vars * sizeof(real)));
    CUDA_CALL(hipMalloc(&d_stimuli, numberOfStimuli * sizeof(Stimulus)));

    CUDA_CALL(hipMemcpy(d_Vm, Vm, total_points * sizeof(real), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_sV, sV, total_points * cell_model_solver->n_state_vars * sizeof(real), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_stimuli, stimuli, numberOfStimuli * sizeof(Stimulus), hipMemcpyHostToDevice));
    
    // Auxiliary variables for the operations
    real *d_RHS;
    CUDA_CALL(hipMalloc(&d_RHS, total_points * sizeof(real)));
    
    // Calculate coefficients for the ADI method
    const real phi_x = delta_t / (delta_x * delta_x);
    const real phi_y = delta_t / (delta_y * delta_y);
    const real diff_coeff = compute_diffusion_coefficient(sigma);
    
    // CUDA grid and block allocation
    // Device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    // Number of SMs and minimum number of blocks to maximize the parallelism
    const int numSMs = prop.multiProcessorCount;
    const int minBlocks = 2 * numSMs;

    // Print information
    SIMPLEMSG("");
    INFOMSG("Device name: %s (%d SMs)\n", prop.name, numSMs);

    // Calculate the number of blocks and threads for the full domain kernels
    dim3 fullDomainBlockSize(FULL_DOMAIN_BLOCK_SIZE_X, FULL_DOMAIN_BLOCK_SIZE_Y);
    dim3 fullDomainGridSize((Nx + fullDomainBlockSize.x - 1) / fullDomainBlockSize.x, (Ny + fullDomainBlockSize.y - 1) / fullDomainBlockSize.y);

    // Adjust the number of blocks
    if (fullDomainGridSize.x * fullDomainGridSize.y < minBlocks)
        fullDomainGridSize.x = (minBlocks + fullDomainGridSize.y - 1) / fullDomainGridSize.y;
    
    // Print information
    SIMPLEMSG("");
    INFOMSG("For full domain kernels:\n");
    INFOMSG("Block size: %d x %d threads (total %d threads per block)\n", fullDomainBlockSize.x, fullDomainBlockSize.y, fullDomainBlockSize.x * fullDomainBlockSize.y);
    INFOMSG("Grid size: %d x %d blocks (total %d blocks, total %d threads)\n", fullDomainGridSize.x, fullDomainGridSize.y, fullDomainGridSize.x * fullDomainGridSize.y, fullDomainGridSize.x * fullDomainGridSize.y * fullDomainBlockSize.x * fullDomainBlockSize.y);

    // Variables for time measurement
    real startTime = 0.0f;
    real startExecutionTime = 0.0f;
    real elapsedExecutionTime = 0.0f;
    real elapsedTime1stPart = 0.0f;
    real elapsedTime2ndPart = 0.0f;
    real elapsedSaveFramesTime = 0.0f;
    real elapsedMeasureVelocityTime = 0.0f;

    SIMPLEMSG("");
    INFOMSG("Starting simulation with FE (CUDA)...\n");

    // Main time loop
    startExecutionTime = omp_get_wtime();

    while (timeStepCounter < M)
    {
        // Get time step
        actualTime = time_array[timeStepCounter];

        // ================================================!
        //  Calculate Approxs. and Update ODEs             !
        // ================================================!
        startTime = omp_get_wtime();

        // Launch kernel to compute the reaction term and update state variables
        solveAndUpdateSV<<<fullDomainGridSize, fullDomainBlockSize>>>(Nx, Ny, delta_t, phi_x, phi_y, diff_coeff, actualTime,
                                                                      numberOfStimuli, d_stimuli, d_Vm, d_sV, d_RHS, cell_model);
        CUDA_CALL(hipDeviceSynchronize());

        elapsedTime1stPart += omp_get_wtime() - startTime;

        // ==================!
        //  Update Vm        !
        // ==================!
        startTime = omp_get_wtime();
        CUDA_CALL(hipMemcpy(d_Vm, d_RHS, total_points * sizeof(real), hipMemcpyDeviceToDevice));
        elapsedTime2ndPart += omp_get_wtime() - startTime;

        // Save frame if needed
        if (saveFrames && (timeStepCounter % frameSaveRate == 0))
        {
            startTime = omp_get_wtime();
            CUDA_CALL(hipMemcpy(Vm, d_Vm, total_points * sizeof(real), hipMemcpyDeviceToHost));
            handle_frame_saving(pathToSaveData, file_extension, save_function, timeStepCounter, Vm, Nx, Ny, delta_x, delta_y, actualTime);
            elapsedSaveFramesTime += omp_get_wtime() - startTime;
        }

        // Measure velocity if needed
        if (measureVelocity && !stim_velocity_measured)
        {
            startTime = omp_get_wtime();

            // Copy only the Vm values needed for velocity measurement
            real Vmidx_x0, Vmidx_x1;
            CUDA_CALL(hipMemcpy(&Vmidx_x0, &d_Vm[idx_x0], sizeof(real), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(&Vmidx_x1, &d_Vm[idx_x1], sizeof(real), hipMemcpyDeviceToHost));
            handle_velocity_measurement(Vmidx_x0, Vmidx_x1, &t0, &t1, activation_threshold, &aux_stim_velocity_flag, &stim_velocity_measured, actualTime, x0, x1, &stim_velocity);
            
            elapsedMeasureVelocityTime += omp_get_wtime() - startTime;
        }
        
        // Update time step counter
        timeStepCounter++;
    }

    elapsedExecutionTime = omp_get_wtime() - startExecutionTime;

    INFOMSG("Simulation done!\n");
    SIMPLEMSG("");

    // Update measurement structure
    measurement->elapsedExecutionTime = elapsedExecutionTime;
    measurement->elapsedTime1stPart = elapsedTime1stPart;
    measurement->elapsedTime2ndPart = elapsedTime2ndPart;
    measurement->elapsedSaveFramesTime = elapsedSaveFramesTime;
    measurement->elapsedMeasureVelocityTime = elapsedMeasureVelocityTime;
    measurement->stimVelocity = stim_velocity;

    // Copy results back to host
    CUDA_CALL(hipMemcpy(Vm, d_Vm, total_points * sizeof(real), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(sV, d_sV, total_points * cell_model_solver->n_state_vars * sizeof(real), hipMemcpyDeviceToHost));

    // Free allocated memory
    CUDA_CALL(hipFree(d_Vm));
    CUDA_CALL(hipFree(d_sV));
    CUDA_CALL(hipFree(d_stimuli));
    CUDA_CALL(hipFree(d_RHS));
}
