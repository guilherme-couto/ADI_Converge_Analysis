#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    hipDeviceProp_t prop;
    int deviceCount;
    
    hipGetDeviceCount(&deviceCount);
    if (deviceCount > 0) {
        hipGetDeviceProperties(&prop, 0);
        printf("Device name: %s\n", prop.name);
    } else {
        printf("No CUDA devices found.\n");
    }

    return 0;
}
